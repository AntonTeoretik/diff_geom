#include "hip/hip_runtime.h"
#include "ellipsoid.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

using namespace std;

#include "algebra.h"
#include "metric.h"
#include "graphics.h"
#include "color.h"

#include <fstream>

#include <array>


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


class my_test_funtion_on_manifold : public genFunction<3, Vec<3>>
{
    Ellipsoid<3> S;

    CUDA_F double closeness_to_int(double x) const {
        double x1 = std::round(x);
        x1 = abs(x1 - x);
        return x1 <= 0.1 ? 2.0 : 0.0;
    }
    // genFunction interface
public:
    CUDA_F my_test_funtion_on_manifold(const Ellipsoid<3>& S) : S(S) {}

    CUDA_F Vec<3> operator ()(const genPoint<3> & p) const
    {
        Point<3> pp = p.p;

        if(p.i == 1 and !S.changePointIndex(pp, 1, 0)) {
            return Vec<3>::zero();
        }
        //pp.scale(0.1);

        // double x = closeness_to_int( pp[0] - 0.5);
        // double y = closeness_to_int( pp[1] - 0.5);
        // double z = closeness_to_int( pp[2] - 0.5);

        // return Vec<3>({x*y, y*z, z*x});

        pp.scale(4);

        int res = int(0.1 - abs(pp[0]) > 0) + int(0.1 - abs(pp[1]) > 0) + int(0.1 - abs(pp[2]) > 0);
        if (res < 1) {
            return {0.0, 0.0, 0.0};
        }
        double x = closeness_to_int(pp[0]);
        double y = closeness_to_int(pp[1]);
        double z = closeness_to_int(pp[2]);
        //double z = std::max(0.0, 0.1 - abs(pp[2]));

        return Vec<3>{x*y, y*z, z*x}*3.6;
    }
};

__global__ void render_kernel(double* out_img, int width, int height, int block_size) {
    int task_idx = threadIdx.x+blockIdx.x*blockDim.x;
    
    // Task indexing
    int nb_h = height / block_size;
    int nb_w = width / block_size;

    if (task_idx >= nb_h * nb_w) {
        return;
    }

    int yb = task_idx / nb_w;
    int xb = task_idx % nb_w;
    
    int x1 = xb * block_size;
    int x2 = (xb + 1) * block_size;

    int y1 = yb * block_size;
    int y2 = (yb + 1) * block_size;

    // Setup env
    Ellipsoid<3> S(Vec<4>({1.0, 1.0, 1.0, 1.0}), 9.0);
    my_test_funtion_on_manifold f(S);

    standartWeight w(0.2);

    //auto metr = EllipsoidMetric<3>({1,1,1,1}, NORTH);
    //std::cout << "Metr " << metr({0.0, 0.0, 0.0}, {1.0, 0.0, 0.0}, {1.0, 0.0, 0.0}) << std::endl ;

    Renderer rend(S, f, {0, {5,5,1}}, w);

    rend.number_of_points = 330;
    rend.step = 0.01;
    rend.screen.h_resolution = height;
    rend.screen.w_resolution = width;
    //1366x768

    rend.screen.height = 7.68;
    rend.screen.width = 13.66;
    rend.screen.distance_to_screen = 3.0;
    rend.setBasis(Vec<3>({-5, -5, -1}), Vec<3>({0, 0, -1}), Vec<3>({0, 1, 0}));

    // Do the work
    for (int x = x1; x < x2; ++x) {
        for (int y = y1; y < y2; ++y) {
            auto res = rend.trace_one_pixel(x, y);

            int offset = (y * width + x) * 3;
            out_img[offset] = res[0];
            out_img[offset + 1] = res[1];
            out_img[offset + 2] = res[2];
        }
    }
}

void test_one() {
    int width = 100;
    int height = 100;
    int y = 50;
    int x = 50;

    Ellipsoid<3> S(Vec<4>({1.0, 1.0, 1.0, 1.0}), 9.0);
    my_test_funtion_on_manifold f(S);
    standartWeight w;

    //auto metr = EllipsoidMetric<3>({1,1,1,1}, NORTH);
    //std::cout << "Metr " << metr({0.0, 0.0, 0.0}, {1.0, 0.0, 0.0}, {1.0, 0.0, 0.0}) << std::endl ;

    Renderer rend(S, f, {0, {0,0,0}}, w);

    rend.number_of_points = 500;
    rend.step = 0.01;
    rend.screen.h_resolution = width;
    rend.screen.w_resolution = height;
    rend.setBasis({1,3,2}, {3,2,1}, {1, 0, 0});

    auto res = rend.trace_one_pixel(x, y);
    printf("%f %f %f\n", res[0], res[1], res[2]);
}

int main()
{
    // test_one();
    // return 0;
    
    int width = 1280;
    int height = 720;
    
    thrust::device_vector<double> pixels(width * height * 3);
    gpuErrchk( hipDeviceSynchronize() );

    // Set stack limit
    auto error = hipDeviceSetLimit(hipLimitStackSize, 16 * 1024);
    if (error != hipSuccess) {
        printf("hipDeviceSetLimit failed with %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    size_t p_val;
    error = hipDeviceGetLimit(&p_val, hipLimitStackSize);
    if (error != hipSuccess) {
        printf("hipDeviceGetLimit failed with %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    printf("stack size limit: %lu\n", p_val);
    
    int bs = 8;

    int block_dim = 32;
    int grid_dim = ((width / bs) * (height / bs)) / block_dim + 1;

    double* raw_gpu_pixels = thrust::raw_pointer_cast( pixels.data() );

    render_kernel<<<grid_dim, block_dim>>>(raw_gpu_pixels, width, height, bs);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    thrust::host_vector<double> pixels_host = pixels;
    gpuErrchk( hipDeviceSynchronize() );

    bitmap_image img(width, height);

    for(size_t x = 0; x < width; x++) {
        for(size_t y = 0; y < height; y++) {
            int offset = (y * width + x) * 3;
            Vec<3> v;
            v[0] = pixels_host[offset];
            v[1] = pixels_host[offset + 1];
            v[2] = pixels_host[offset + 2];
            img.set_pixel(x, y, Color(v).toRGB());
        }
    }
    img.save_image("img.bmp");


    return 0;
}
