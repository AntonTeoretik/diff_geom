#include "hip/hip_runtime.h"
#include "metric.h"



template<std::size_t N>
CUDA_F MetricTensor<N>::MetricTensor()
{
    for(std::size_t i = 0; i < N; i++) {
        for(std::size_t j = 0; j < N; j++) {
            _basis[i][j] = (i == j ? 1.0 : 0.0);
        }
    }
}

template<std::size_t N>
CUDA_F double MetricTensor<N>::call_array(const Point<N>& p, const Vec<N>* vecs) const
{
    return (*this)(p, vecs[0], vecs[1]);
}


template<std::size_t N>
CUDA_F Matrix2D<N> MetricTensor<N>::getMatrix(const Point<N>& P) const
{
    Matrix2D<N> m;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            m.get(i, j) = getCoord(P, i, j);
        }
    }
    return m;
}

template<std::size_t N>
CUDA_F double MetricTensor<N>::getCoord(const Point<N>& P, std::size_t i, std::size_t j) const
{
    // printf("GetCoord %lu %lu %lu\n", N, i, j);
    // return 0.0;
    return (*this)(P, _basis[i], _basis[j]);
}

// template<std::size_t N>
// CUDA_F double MetricTensor<N>::krist(std::size_t l, std::size_t j, std::size_t k, const Point<N>& p) const
// {
//     double res = 0;
//     return 0;

//     // Matrix2D<N> MgInv = getMatrix(p).inverse();

//     // for(std::size_t r = 0; r < N; r++) {
//     //     double dk_grj = partialDer<N>(p, k, [this, r, j] __device__ (const Point<N>& pp){return this->getCoord(pp, r, j);} );
//     //     double dj_grk = partialDer<N>(p, j, [this, r, k] __device__ (const Point<N>& pp){return this->getCoord(pp, r, k);} );
//     //     double dr_gjk = partialDer<N>(p, r, [this, k, j] __device__ (const Point<N>& pp){return this->getCoord(pp, j, k);} );

//     //     res += MgInv.get(l, r) * ( dk_grj + dj_grk - dr_gjk );
//     // }
//     // return res * 0.5;
// }

template<std::size_t N>
CUDA_F Matrix2D<N> MetricTensor<N>::kristMatrix(std::size_t l, const Point<N>& p) const
{
    Matrix2D<N> res;
    // return res;
    for (size_t i = 1; i < N; i++) {
        for (size_t j = 0; j < i; j++) {
            double kr = krist(l, i, j, p);
            res.get(i, j) = kr;
            res.get(j, i) = kr;
        }
    }
    for (size_t i = 0; i < N; i++) {
        res.get(i, i) = this->krist(l, i, i, p);
    }

    //return Matrix2D<N>([this, l, &p](auto i, auto j){return this->krist(l, i, j, p);});
    return res;
}


template<std::size_t N>
CUDA_F void MetricTensor<N>::orthogonalize(
    const Point<N>& pt, const Span<Vec<N>>& vecs, bool norm, 
    Span<Vec<N>>& res, int& n_out) const
{
    n_out = 0;
    // std::vector<Vec<N> > res = {};
    for (size_t i = 0; i < vecs.size; i++) {
        Vec<N> vec = vecs.data[i];
        for(size_t j = 0; j < n_out; j++) {
            vec = vec - res[j] * ((*this)(pt, vecs[i], res[j]) / (*this)(pt, res[j], res[j]) );
        }
        res.data[n_out] = vec;
        ++n_out;
    }
    if (norm) {
        for(int i = 0; i < n_out; ++i) {
            //std::cout << "Scale: " << std::sqrt((*this)(pt, v, v)) << std::endl;
            //std::cout << "v: " << v.to_str() << std::endl;
            auto v = res.data[i];
            res.data[i].scale(1.0 / std::sqrt((*this)(pt, v, v)));
        }
    }
}


template class MetricTensor<1>;
template class MetricTensor<2>;
template class MetricTensor<3>;


template<std::size_t N, std::size_t M>
CUDA_F InducedMetricTensor<N, M>::InducedMetricTensor(double pres) :
    MetricTensor<N>(),
    pres(pres),
    inv_pres2(0.25 / (pres * pres)),
    inv_pres3(0.125 / (pres * pres * pres))
{
    //std::cout << pres << " " << inv_pres2 << " " << inv_pres3 << std::endl;
}


template<std::size_t N, std::size_t M>
CUDA_F double InducedMetricTensor<N, M>::operator()(const Point<N> & p, const Vec<N> & v1_, const Vec<N> & v2_) const
{
    Vec<N> v1 = v1_;
    Vec<N> v2 = v2_;
    double alpha1 = v1.norm();
    double alpha2 = v2.norm();
    v1.normalize();
    v2.normalize();
    v1.scale(pres);
    v2.scale(pres);

    Vec<M> dv1p, dv1m, dv2p, dv2m;

    if(alpha1 * alpha2 == 0) return 0.0; // Almost impossible, but anyway

    dv1p = gen_func(p + v1);
    dv1m = gen_func(p - v1);
    dv2p = gen_func(p + v2);
    dv2m = gen_func(p - v2);
    dv1p.subtract(dv1m);
    dv2p.subtract(dv2m);

    return alpha1 * alpha2 * (dv1p * dv2p) * 0.25 / pres / pres;
};

template<std::size_t N, std::size_t M>
CUDA_F double InducedMetricTensor<N, M>::dk_gij(const Point<N> &p, size_t k, size_t i, size_t j) const
{
    //partialDer<N>(pt, k, [this, i, j](Point<N>& pp){return this->getCoord(pp, i, j);} );

    Point<N> pt = p;

    double pt_k = pt[k];
    double pt_i = pt[i];
    double pt_j = pt[j];

    Vec<M> vec_pxi, vec_mxi, vec_pxj, vec_mxj;

    pt[i] += pres; pt[k] += pres;
    vec_pxi = gen_func(pt);
    pt[i] = pt_i; pt[k] = pt_k;

    pt[i] -= pres; pt[k] += pres;
    vec_mxi = gen_func(pt);
    pt[i] = pt_i; pt[k] = pt_k;

    pt[j] += pres; pt[k] += pres;
    vec_pxj = gen_func(pt);
    pt[j] = pt_j; pt[k] = pt_k;

    pt[j] -= pres; pt[k] += pres;
    vec_mxj = gen_func(pt);
    pt[j] = pt_j; pt[k] = pt_k;

    vec_pxi.subtract(vec_mxi);
    vec_pxj.subtract(vec_mxj);

    double subres1 = vec_pxi * vec_pxj;


    pt[i] += pres; pt[k] -= pres;
    vec_pxi = gen_func(pt);
    pt[i] = pt_i; pt[k] = pt_k;

    pt[i] -= pres; pt[k] -= pres;
    vec_mxi = gen_func(pt);
    pt[i] = pt_i; pt[k] = pt_k;

    pt[j] += pres; pt[k] -= pres;
    vec_pxj = gen_func(pt);
    pt[j] = pt_j; pt[k] = pt_k;

    pt[j] -= pres; pt[k] -= pres;
    vec_mxj = gen_func(pt);
    pt[j] = pt_j; pt[k] = pt_k;

    vec_pxi.subtract(vec_mxi);
    vec_pxj.subtract(vec_mxj);

    double subres2 = vec_pxi * vec_pxj;

    return (subres1 - subres2) * 0.125 / pres / pres / pres;
}

template<std::size_t N, std::size_t M>
CUDA_F double InducedMetricTensor<N, M>::krist(std::size_t l, std::size_t j, std::size_t k, const Point<N> &p) const
{
    double res = 0;
    // double x = getCoord(p, 0, 0);;
    // return res;

    Matrix2D<N> Mg;
    for(size_t i = 0; i < N; i++) {
        for(size_t jj = 0; jj < N; jj++) {
            Mg.get(i, jj) = getCoord(p, i, jj);
        }
    }
    //Matrix2D<N> MgInv = MetricTensor<N>::getMatrix(p).inverse();
    Matrix2D<N> MgInv = Mg.inverse();

    for(std::size_t r = 0; r < N; r++) {
        double dk_grj = dk_gij(p, k, r, j);
        double dj_grk = dk_gij(p, j, k, r);
        double dr_gjk = dk_gij(p, r, j, k);

        res += MgInv.get(l, r) * ( dk_grj + dj_grk - dr_gjk );
    }
    return res * 0.5;
}

template<std::size_t N, std::size_t M>
CUDA_F double InducedMetricTensor<N, M>::getCoord(const Point<N> &p, std::size_t i, std::size_t j) const
{
    // printf("GetCoord\n");
    Point<N> pt = p;
    double pt_i = pt[i];
    double pt_j = pt[j];

    Vec<M> vec_pxi, vec_mxi, vec_pxj, vec_mxj;

    pt[i] += pres;
    vec_pxi = gen_func(pt);
    pt[i] = pt_i;

    pt[i] -= pres;
    vec_mxi = gen_func(pt);
    pt[i] = pt_i;

    pt[j] += pres;
    vec_pxj = gen_func(pt);
    pt[j] = pt_j;

    pt[j] -= pres;
    vec_mxj = gen_func(pt);
    pt[j] = pt_j;

    vec_pxi.subtract(vec_mxi);
    vec_pxj.subtract(vec_mxj);
    double subres = vec_pxi * vec_pxj;
    return 0.25 * subres / pres / pres;
}

template class InducedMetricTensor<1,2>;
template class InducedMetricTensor<2,3>;
template class InducedMetricTensor<3,4>;
template class InducedMetricTensor<2,2>;
