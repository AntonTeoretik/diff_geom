#include "hip/hip_runtime.h"
#include "algebra.h"


template<std::size_t N>
std::string Vec<N>::to_str() const
{
    std::string res = "";
    for (int i = 0; i < N; ++i)
        res += std::to_string(data[i]) + " ";
    return res;
}

template<std::size_t N>
CUDA_F Vec<N> Vec<N>::operator+(const Vec<N> &B) const
{
    Vec<N> res;
    for(std::size_t i = 0; i < N; i++)
        res[i] = data[i] + B[i];
    return res;
}

template<std::size_t N>
CUDA_F Vec<N> Vec<N>::operator-(const Vec<N> &B) const
{
    Vec<N> res;
    for(std::size_t i = 0; i < N; i++)
        res[i] = data[i] - B[i];
    return res;
}

template<std::size_t N>
CUDA_F Vec<N> Vec<N>::operator*(const double &a) const
{
    Vec<N> res;
    for(std::size_t i = 0; i < N; i++)
        res[i] = a * data[i];
    return res;
}

template<std::size_t N>
CUDA_F double Vec<N>::operator*(const Vec<N> &v) const
{
    double res = 0;
    for(std::size_t i = 0; i < N; i++)
        res += data[i] * v[i];
    return res;
}

template<std::size_t N>
CUDA_F double Vec<N>::norm2() const
{
    double res = 0;
    for (int i = 0; i < N; ++i) {
        double c = data[i];
        res += c*c;
    }
    return res;
}

template<std::size_t N>
CUDA_F double Vec<N>::norm() const
{
    return sqrt(norm2());
}

template<std::size_t N>
CUDA_F Vec<N> Vec<N>::normalized() const
{
    return (*this) * (1 / norm());
}

template<std::size_t N>
CUDA_F void Vec<N>::normalize()
{
    double inv_norm = 1.0 / this->norm();
    for(std::size_t i = 0; i < N; i++)
        data[i] *= inv_norm;
}

template<std::size_t N>
CUDA_F void Vec<N>::addWithScale(const Vec<N> &v, double scale)
{
    for(std::size_t i = 0; i < N; i++)
        data[i] += v[i] * scale;
}

template<std::size_t N>
CUDA_F void Vec<N>::subtract(const Vec<N> &v)
{
    for(std::size_t i = 0; i < N; i++)
        data[i] -= v[i];
}

template<std::size_t N>
CUDA_F void Vec<N>::scale(double a)
{
    for(std::size_t i = 0; i < N; i++)
        data[i] *= a;
}

template<std::size_t N>
CUDA_F Vec<N> Vec<N>::zero()
{
    Vec<N> res;
    for(std::size_t i = 0; i < N; i++)
        res[i] = 0;
    return res;
}



template<>
CUDA_F Matrix2D<1> Matrix2D<1>::inverse() const
{
    Matrix2D<1> mat;
    mat.get(0,0) = 1.0 / get(0,0);
    return mat;
}

template<>
CUDA_F Matrix2D<2> Matrix2D<2>::inverse() const
{
    double a = data[0];
    double b = data[1];
    double c = data[2];
    double d = data[3];

    double detinv = 1.0 / (a * d - b * c);

    Matrix2D<2> res;
    res.get(0,0) = d * detinv;
    res.get(0,1) = -b * detinv;
    res.get(1,0) = -c * detinv;
    res.get(1,1) = a * detinv;
    return res;
}


// template<std::size_t N>
// CUDA_F Matrix2D<N>::Matrix2D(std::initializer_list<std::array<double, N> > list)
// {
//     for(std::size_t i = 0; i < N; i++) {
//         (*this)[i] = std::array<double, N>{*(list.begin() + i)};
//     }
// }

// template<std::size_t N>
// Matrix2D<N>::Matrix2D(std::function<double (int, int)> f)
// {
//     for(std::size_t i = 0; i < N; i++) {
//         for(std::size_t j = 0; j < N; j++) {
//             (*this)[i][j] = f(i, j);
//         }
//     }
// }

// template <size_t N>
// CUDA_F Matrix2D<N>::Matrix2D(const NNArray* _data) {
//     for (int i = 0; i < N; ++i) {
//         for (int j = 0; j < N; ++j) {
//             get(i, j) = (*_data)[i][j];
//         }
//     }
// }

template<>
CUDA_F Matrix2D<3> Matrix2D<3>::inverse() const
{
    double a11 = data[0];
    double a12 = data[1];
    double a13 = data[2];

    double a21 = data[3];
    double a22 = data[4];
    double a23 = data[5];

    double a31 = data[6];
    double a32 = data[7];
    double a33 = data[8];

    double detinv = 1.0 / (a11*a22*a33 + a13*a21*a32 + a12*a23*a31 - a13*a22*a31 - a12*a21*a33 - a11*a23*a32);

    Matrix2D<3> adj;
    double inv_data[9] = {
        a22*a33 - a23*a32, -a12*a33 + a13*a32, a23*a12 - a22*a13,
        -a21*a33 + a31*a23,  a11*a33 - a31*a13, -a11*a23 + a21*a13,
        a21*a32 - a22*a31, -a11*a32 + a12*a31,  a22*a11 - a21*a12 
    };
    
    memcpy(adj.data, inv_data, 9 * sizeof(double));
    return adj * detinv;
}

template<std::size_t N>
CUDA_F Vec<N> Matrix2D<N>::operator*(const Vec<N> &A) const
{
    Vec<N> res;
    for(std::size_t i = 0; i < N; i++) {
        double subres = 0.0;
        for(std::size_t j = 0; j < N; j++)
            subres += get(i, j) * A[j];
        res[i] = subres;
    }
    return res;
}

template<std::size_t N>
CUDA_F Matrix2D<N> Matrix2D<N>::operator*(const Matrix2D<N> &A) const
{
    Matrix2D<N> res;
    for(std::size_t i = 0; i < N; i++) {
        for(std::size_t j = 0; j < N; j++) {
            double subres = 0.0;
            for(std::size_t r = 0; r < N; r++) {
                subres += get(i, r) * A.get(r, j);
            }
            res.get(i,j) = subres;
        }
    }

    return res;
}

template<std::size_t N>
CUDA_F Matrix2D<N> Matrix2D<N>::operator*(double a) const
{
    Matrix2D<N> res;
    for(std::size_t i = 0; i < N; i++) {
        for(std::size_t j = 0; j < N; j++) {
            res.get(i, j) = a * get(i, j);
        }
    }

    return res;
}

template<std::size_t N>
std::string Matrix2D<N>::to_str() const
{
    std::string str = "";
    for(std::size_t i = 0; i < N; i++) {
        for(std::size_t j = 0; j < N; j++) {
            str += std::to_string(get(i, j)) + " ";
        }
        str += "\n";
    }
    return str;
}



template class Vec<1>;
template class Vec<2>;
template class Vec<3>;
template class Vec<4>;

template class Matrix2D<1>;
template class Matrix2D<2>;
template class Matrix2D<3>;
template class Matrix2D<4>;



