#include "hip/hip_runtime.h"
#include "manifold.h"



// template<std::size_t N>
// CUDA_F void AbstractRiemannianManifold<N>::geodesic(
//         const genPoint<N> &pt,
//         Vec<N> dir,
//         size_t num_of_pts,
//         size_t dist,
//         double step,
//         Span<genPoint<N>>& res,
//         int& n_out
//         ) const
// {
//     Point<N> prev = pt.p;
//     Point<N> now = prev + (dir * step); // Possible trouble here

//     size_t count = 1;

//     res.data[0] = pt;
//     n_out = 1;
//     if (dist == 1) {
//         res.data[n_out] = genPoint<N>{pt.i, now};
//         count = 0;
//         ++n_out;
//     }

//     chart_index cur_index = pt.i;

//     for (size_t i = 0; i < num_of_pts; i++) {
//         count++;
//         doOneStepWithChange(prev, now, cur_index);

//         if (count == dist) {
//             count = 0;
//             res.data[n_out] = getPoint<N>{cur_index, now};
//             ++n_out;
//         }
//     }
// }


template<std::size_t N>
CUDA_F void AbstractRiemannianManifold<N>::orthogonalize(
    const genPoint<N> &pt, const Span<Vec<N> > &vecs, bool normalize,
    Span<Vec<N>>& out, int& n_out) const
{
    return this->getMetric(pt.i).orthogonalize(pt.p, vecs, normalize, out, n_out);
}


// template<class T, std::size_t N>
// T integrateAlongPath(const std::vector<genPoint<N> > &points,
//                      const std::function<T (genPoint<N>)> &func,
//                      const std::function<double (double)> &weight,
//                      double step)
// {
//     double current_time = 0.0;
//     T res = {};
//     //first value
//     if(points.size() == 0) {
//         return res;
//     }
//     res = func(points[0]) * (step * weight(current_time));

//     for(size_t i = 1; i <= points.size(); i++) {
//         current_time += step;
//         res = res + func(points[i]) * (step * weight(current_time));
//     }
//     return res;
// }

template<size_t N>
template<class T>
CUDA_F T AbstractRiemannianManifold<N>::integrateAlongPath(genPoint<N> start,
                                                  Vec<N> dir, size_t num_of_pts,
                                                  const genFunction<N, T> &func,
                                                  const weightFunction &weight,
                                                  double step) const
{
    //std::cout << "AbstractRiemannianManifold<N>::integrateAlongPath : " << dir.to_str() << std::endl;

    double current_time = 0.0;

    Point<N> prev = start.p;
    Point<N> now = prev + (dir * step); // Possible trouble here

    chart_index cur_index = start.i;

    //NORMALIZE DIR
    double norm2 = getMetric(cur_index)(prev, dir, dir);
    dir.scale(1.0 / sqrt(norm2));

    T res = func(genPoint<N>{cur_index, now}) * weight(0.0) * step;

    for(size_t i = 1; i <= num_of_pts; i++) {
        current_time += step;
        doOneStepWithChange(prev, now, cur_index);
        genPoint<N> pt{cur_index, now};
        res = res + func(pt) * (step * weight(current_time));
    }
    return res;
}

template<std::size_t N>
Point<N> AbstractRiemannianManifold<N>::doOneStep(Point<N> prev, Point<N> now, chart_index i) const
{
    //Try to define next point in this domain.

    Vec<N> curv;
    for (size_t l = 0; l < N; l++) {
        curv[l] = (now - prev) * (getMetric(i).kristMatrix(l, now) * (now - prev));
    }

    auto next = now * 2 - prev - curv;

    return next;
}

template<std::size_t N>
CUDA_F void AbstractRiemannianManifold<N>::doOneStepWithChange(Point<N> &prev, Point<N> &now, chart_index cur_index) const
{
    //std::cout << prev.to_str() << "; " << now.to_str() << "; " << cur_index << std::endl;
    auto next = doOneStep(prev, now, cur_index);

    // next is in current domain
    if(this->isPoint(next, cur_index)) {
        prev = now;
        now = next;
    }
    // next point is outside -> need to change domain.
    else {
        bool change_failed = true;
        for(chart_index new_index = 0; new_index < this->atlas_size; new_index++) {
            if (this->changePointIndex(prev, cur_index, new_index) and
                this->changePointIndex(now, cur_index, new_index) )
            {

                auto alt_next = doOneStep(prev, now, new_index);
                // Everything is good
                if(this->isPoint(alt_next, new_index)) {
                    prev = now;
                    now = alt_next;
                    cur_index = new_index;

                    change_failed = false;
                    break;
                }
            }
        }
        if(change_failed) {
            //std::cout << "Cur_index: " << cur_index << ". " << std::endl;
            //std::cout << now.to_str() << std::endl;
            //std::cout << next.to_str() << std::endl;
            // exit(1);
            // throw std::logic_error("AbstractRiemannianManifold<N>::doOneStepWithChange : failed to change point index. This situation must be impossible, please check configuration of manifold");
        }
    }
}


template
CUDA_F double AbstractRiemannianManifold<3>::integrateAlongPath(genPoint<3>,
                                                  Vec<3>, size_t,
                                                  const genFunction<3, double> &,
                                                  const weightFunction &,
                                                  double step) const;

template
CUDA_F Vec<3> AbstractRiemannianManifold<3>::integrateAlongPath(genPoint<3>,
                                                  Vec<3>, 
                                                  size_t,
                                                  const genFunction<3, Vec<3>> &,
                                                  const weightFunction &,
                                                  double step) const;


template class AbstractRiemannianManifold<3>;
template class AbstractRiemannianManifold<2>;
template class AbstractRiemannianManifold<1>;


